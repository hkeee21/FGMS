#include "spconv.h"
#include "spconv.cuh"
#include "conv_back.cuh"

#include <ATen/ATen.h>
#include <ATen/cuda/HIPContext.h>

#include <math.h>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <torch/extension.h>

#include <hipblas.h>

#include <hip/hip_runtime.h>

#define DIV_UP(x, y) ((x) + (y) - 1) / (y)


void ConvolutionForwardFused(
                        const at::Tensor in_feats, 
                        const at::Tensor kernel, 
                        const int sum_nnz, 
                        at::Tensor out_feats, 
                        const at::Tensor kpos, 
                        const at::Tensor qkpos, 
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const bool separate_mid, 
                        const bool arch80
                        ){
    
  int in_nnz = in_feats.size(0);
  int out_nnz = out_feats.size(0);
  int in_channel = in_feats.size(1);
  if (in_feats.size(1) != kernel.size(1)) {
    throw std::invalid_argument(
        "Input feature size and kernel size mismatch"
        );
  }
  int out_channel = kernel.size(2);
  int k_vol = kernel.size(0);

  bool data_type_half = 
    in_feats.scalar_type() == at::ScalarType::Half;

  int *in_map_ptr = in_map.data_ptr<int>();
  int *out_map_ptr = out_map.data_ptr<int>();
  int mid_weight_id = (k_vol % 2 == 1) ? k_vol / 2 : 0;

  // cublas setup
  const float alpha = 1.0;
  const float beta = 0.0;
  at::Tensor alpha_half = torch::ones({1}, dtype(at::ScalarType::Half));
  at::Tensor beta_half = torch::zeros({1}, dtype(at::ScalarType::Half));

  hipblasComputeType_t ComputeType;
  hipDataType DataType;
  if (data_type_half){
    ComputeType = HIPBLAS_COMPUTE_16F;
    DataType = HIP_R_16F;
  }
  else{
    ComputeType = arch80 ? 
      HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    DataType = HIP_R_32F;
  }

  hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();
  hipblasSetStream(cublasH, 0);
  hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH);

  if (separate_mid){
  // computation for w[0, 0, 0]
  // in_nnz == out_nnz
  if (data_type_half){
    hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
      out_channel, in_nnz, in_channel, 
      reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()), 
      reinterpret_cast<half *>(kernel.data_ptr<at::Half>() 
        + mid_weight_id * in_channel * out_channel),
      DataType, out_channel, 
      reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
      DataType, in_channel, 
      reinterpret_cast<half *>(beta_half.data_ptr<at::Half>()), 
      reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), 
      DataType, out_channel,
      ComputeType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
  }
  else{
    hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
      out_channel, in_nnz, in_channel, 
      &alpha, 
      (kernel.data_ptr<float>() + mid_weight_id * in_channel * out_channel),
      DataType, out_channel, 
      in_feats.data_ptr<float>(), 
      DataType, in_channel, 
      &beta, 
      out_feats.data_ptr<float>(), 
      DataType, out_channel,
      ComputeType, CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
  }

  if (data_type_half){
    if (in_channel % 4 == 0 && out_channel % 4 == 0){   
      if (in_channel <= 16 || out_channel <= 16){
        _fgms_fusion_fp16_4_once<16, 4, 8>
            <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1), dim3(4, 16, 1)>>>(
            kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
            reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
            reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
            reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
            in_map_ptr, out_map_ptr
        );
      }
      else{
        if (arch80){
          _fgms_fusion_fp16_tc4_async<32, 4, 8, 16, 16, 16, 4, 2, 2>
              <<<dim3(DIV_UP(out_channel, 32), DIV_UP(sum_nnz, 128), 1), dim3(8, 32, 1)>>>(
              kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
              reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
              reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
              reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
              in_map_ptr, out_map_ptr
          );
        }
        else{
          _fgms_fusion_fp16_tc4<32, 4, 8, 16, 16, 16, 4, 2, 2>
              <<<dim3(DIV_UP(out_channel, 32), DIV_UP(sum_nnz, 128), 1), dim3(8, 32, 1)>>>(
              kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
              reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
              reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
              reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
              in_map_ptr, out_map_ptr
          );
        }
      }
    }
    else if (in_channel % 2 == 0 && out_channel % 2 ==0){
      _fgms_fusion_fp16_2<16, 8, 8><<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 128), 1), dim3(8, 16, 1)>>>(
          kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
          reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
          reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
          reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
          in_map_ptr, out_map_ptr
      );   
    }
    else{
      _fgms_fusion_fp16_1<16, 4, 8><<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1), dim3(16, 16, 1)>>>(
          kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
          reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()),
          reinterpret_cast<half *>(kernel.data_ptr<at::Half>()),
          reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()),
          in_map_ptr, out_map_ptr
      );   
    }  
  }
  else{
    if(in_channel % 4 == 0 && out_channel % 4 == 0){
      if (in_channel <= 16 && out_channel <= 16){
        _fgms_fusion_fp32_once<16, 4, 8>
            <<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1), dim3(4, 16, 1)>>>(
            kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
            in_feats.data_ptr<float>(), kernel.data_ptr<float>(), out_feats.data_ptr<float>(), 
            in_map_ptr, out_map_ptr
        );
      }
      else{
        if (arch80){
          _fgms_fusion_tf32<32, 4, 8, 16, 8, 16, 4, 2, 2>
              <<<dim3(DIV_UP(out_channel, 32), DIV_UP(sum_nnz, 128), 1), dim3(8, 32, 1)>>>(
              kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
              in_feats.data_ptr<float>(), kernel.data_ptr<float>(), out_feats.data_ptr<float>(), 
              in_map_ptr, out_map_ptr
          );
        }
        else{
          _fgms_fusion_fp32<32, 4, 8>
              <<<dim3(DIV_UP(out_channel, 32), DIV_UP(sum_nnz, 128), 1), dim3(8, 32, 1)>>>(
              kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
              in_feats.data_ptr<float>(), kernel.data_ptr<float>(), out_feats.data_ptr<float>(), 
              in_map_ptr, out_map_ptr
          );
        }
      }
    }
    else if (in_channel % 2 == 0){
      _fgms_fusion_fp32_2<16, 8, 8><<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 128), 1), dim3(8, 16, 1)>>>(
          kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
          in_feats.data_ptr<float>(), kernel.data_ptr<float>(), out_feats.data_ptr<float>(), 
          in_map_ptr, out_map_ptr
      );
    }
    else{
      _fgms_fusion_fp32_1<16, 4, 8><<<dim3(DIV_UP(out_channel, 16), DIV_UP(sum_nnz, 64), 1), dim3(16, 16, 1)>>>(
          kpos.data_ptr<int>(), qkpos.data_ptr<int>(), k_vol, in_channel, out_channel, 
          in_feats.data_ptr<float>(), kernel.data_ptr<float>(), out_feats.data_ptr<float>(), 
          in_map_ptr, out_map_ptr
      );
    }
  }
}


void ConvolutionForward(const at::Tensor in_feats, 
                        const at::Tensor kernel, 
                        const int sum_nnz, 
                        at::Tensor out_feats, 
                        const at::Tensor kernel_nnz, 
                        const at::Tensor kernel_pos, 
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const bool separate_mid, 
                        const bool arch80
                        ){

  int in_nnz = in_feats.size(0);
  int out_nnz = out_feats.size(0);
  int in_channel = in_feats.size(1);
  if (in_feats.size(1) != kernel.size(1)) {
    throw std::invalid_argument(
        "Input feature size and kernel size mismatch"
        );
  }
  int out_channel = kernel.size(2);
  int k_vol = kernel.size(0);

  bool data_type_half = 
    in_feats.scalar_type() == at::ScalarType::Half;

  int *in_map_ptr = in_map.data_ptr<int>();
  int *out_map_ptr = out_map.data_ptr<int>();
  int *kpos_ptr = kernel_pos.data_ptr<int>();

  int mid_weight_id = (k_vol % 2 == 1) ? k_vol / 2 : 0;

  // cublas setup
  const float alpha = 1.0;
  const float beta = 1.0;
  at::Tensor alpha_half = torch::ones({1}, dtype(at::ScalarType::Half));
  at::Tensor beta_half = torch::ones({1}, dtype(at::ScalarType::Half));

  hipblasComputeType_t ComputeType;
  hipDataType DataType;
  if (data_type_half){
    ComputeType = HIPBLAS_COMPUTE_16F;
    DataType = HIP_R_16F;
  }
  else{
    ComputeType = arch80 ? 
      HIPBLAS_COMPUTE_32F_FAST_TF32 : HIPBLAS_COMPUTE_32F;
    DataType = HIP_R_32F;
  }

  hipblasHandle_t cublasH = at::cuda::getCurrentCUDABlasHandle();
  hipblasSetStream(cublasH, 0);
  hipblasSetMathMode(cublasH, HIPBLAS_TENSOR_OP_MATH);

  /********************************************************************/
  // loop over all kernel offsets
  int cur_idx = 0;
  // int stream_id = 0;
  for (int k = 0; k < k_vol; k++){
    int cur_nnz = kernel_nnz.data_ptr<int>()[k];
    
    if (cur_nnz == 0){continue;}

    size_t gridnum_x = DIV_UP(out_channel, 32);
    size_t gridnum_y = DIV_UP(cur_nnz, 32);

    if (data_type_half){
      if (in_channel % 4 == 0 && out_channel % 4 == 0){
        _fgms_seq_fp16<32, 4, 8, 16, 16, 16, 4, 2, 2>
            <<<dim3(gridnum_x, gridnum_y, 1), dim3(8, 32, 1)>>>(
            cur_nnz, in_channel, out_channel, 
            reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
            reinterpret_cast<half *>(kernel.data_ptr<at::Half>() 
                + k * in_channel * out_channel), 
            reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), 
            &in_map_ptr[cur_idx], &out_map_ptr[cur_idx]
        );
      }
      else{
        _fgms_seq_fp16_1<16, 4, 8>
            <<<dim3(DIV_UP(out_channel, 16), DIV_UP(cur_nnz, 16), 1), dim3(16, 16, 1)>>>(
            cur_nnz, in_channel, out_channel, 
            reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
            reinterpret_cast<half *>(kernel.data_ptr<at::Half>() 
                + k * in_channel * out_channel), 
            reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), 
            &in_map_ptr[cur_idx], &out_map_ptr[cur_idx]
        );
      }
    }
    else{
      if (in_channel % 4 == 0 && out_channel % 4 == 0){
        if (arch80){
          _fgms_seq_tf32<32, 4, 8, 16, 8, 16, 4, 2, 2>
              <<<dim3(gridnum_x, gridnum_y, 1), dim3(8, 32, 1)>>>(
              cur_nnz, in_channel, out_channel, 
              in_feats.data_ptr<float>(), 
              (kernel.data_ptr<float>() + k * in_channel * out_channel), 
              out_feats.data_ptr<float>(), 
              &in_map_ptr[cur_idx], &out_map_ptr[cur_idx]
          );
        }
        else{
          _fgms_seq_fp32<32, 4, 8>
              <<<dim3(gridnum_x, gridnum_y, 1), dim3(8, 32, 1)>>>(
              cur_nnz, in_channel, out_channel, 
              in_feats.data_ptr<float>(), 
              (kernel.data_ptr<float>() + k * in_channel * out_channel), 
              out_feats.data_ptr<float>(), 
              &in_map_ptr[cur_idx], &out_map_ptr[cur_idx]
          );
        }
      }
      else{
        _fgms_seq_fp32_1<16, 4, 8>
            <<<dim3(DIV_UP(out_channel, 16), DIV_UP(cur_nnz, 16), 1), dim3(16, 16, 1)>>>(
            cur_nnz, in_channel, out_channel, 
            in_feats.data_ptr<float>(), 
            (kernel.data_ptr<float>() + k * in_channel * out_channel), 
            out_feats.data_ptr<float>(), 
            &in_map_ptr[cur_idx], &out_map_ptr[cur_idx]
        );
      }
    }

    cur_idx += cur_nnz;
  }

  // put behind to avoid AtomicAdd in GMS kernels
  if (separate_mid){
  // computation for w[0, 0, 0]
  // in_nnz == out_nnz
    if (data_type_half){
      hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        out_channel, in_nnz, in_channel, 
        reinterpret_cast<half *>(alpha_half.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(kernel.data_ptr<at::Half>() 
            + mid_weight_id * in_channel * out_channel),
        DataType, out_channel, 
        reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
        DataType, in_channel, 
        reinterpret_cast<half *>(beta_half.data_ptr<at::Half>()), 
        reinterpret_cast<half *>(out_feats.data_ptr<at::Half>()), 
        DataType, out_channel,
        ComputeType,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
    else{
      hipblasGemmEx(cublasH, HIPBLAS_OP_N, HIPBLAS_OP_N, 
        out_channel, in_nnz, in_channel, 
        &alpha, 
        (kernel.data_ptr<float>() + mid_weight_id * in_channel * out_channel),
        DataType, out_channel, 
        in_feats.data_ptr<float>(), 
        DataType, in_channel, 
        &beta, 
        out_feats.data_ptr<float>(), 
        DataType, out_channel,
        ComputeType,
        CUBLAS_GEMM_DEFAULT_TENSOR_OP);
    }
  }
}


void ConvolutionBackward(const at::Tensor out_feats_grad, 
                        const at::Tensor in_feats, 
                        const at::Tensor kernel, 
                        const int sum_nnz, 
                        at::Tensor in_feats_grad, 
                        at::Tensor kernel_grad, 
                        const at::Tensor kpos,
                        const at::Tensor qkpos, 
                        const at::Tensor in_map, 
                        const at::Tensor out_map, 
                        const bool separate_mid,
                        const bool arch80
                        ){
  
  int innz = in_feats.size(0);
  int onnz = out_feats_grad.size(0);
  int in_channel = in_feats.size(1);
  if (in_feats.size(1) != kernel.size(1)) {
    throw std::invalid_argument(
      "Input feature size and kernel size mismatch"
    );
  }
  int out_channel = kernel.size(2);
  int k_vol = kernel.size(0);

  bool data_type_half = 
    in_feats.scalar_type() == at::ScalarType::Half;

  int *in_map_ptr = in_map.data_ptr<int>();
  int *out_map_ptr = out_map.data_ptr<int>();

  int *kpos_ptr = kpos.data_ptr<int>();
  int *qkpos_ptr = qkpos.data_ptr<int>();

  int mid_weight_id = (k_vol % 2 == 1) ? k_vol / 2 : 0;

  // loop over all kernel offsets: 
  // W^T X {\delta{out_feats}} = {\delta{in_feats}}^T
  // {\delta{out_feats}}^T X in_feats = {\delta{W}}^T
  if (data_type_half){
    _fgms_fusion_fp16_W_transpose<32, 4, 8, 16, 16, 16, 4, 2, 2>
              <<<dim3(DIV_UP(in_channel, 32), DIV_UP(sum_nnz, 128), 1), dim3(4, 32, 1)>>>(
                kpos_ptr, qkpos_ptr, k_vol, in_channel, out_channel, 
                reinterpret_cast<half *>(out_feats_grad.data_ptr<at::Half>()), 
                reinterpret_cast<half *>(kernel.data_ptr<at::Half>()), 
                reinterpret_cast<half *>(in_feats_grad.data_ptr<at::Half>()), 
                out_map_ptr, in_map_ptr
          );
    _fgms_fusion_fp16_I_transpose<32, 4, 8, 16, 16, 16, 2, 2, 1>
              <<<dim3(DIV_UP(sum_nnz, 128)), dim3(4, 32, 1)>>>(
                kpos_ptr, qkpos_ptr, k_vol, in_channel, out_channel, 
                reinterpret_cast<half *>(in_feats.data_ptr<at::Half>()), 
                reinterpret_cast<half *>(out_feats_grad.data_ptr<at::Half>()),
                reinterpret_cast<half *>(kernel_grad.data_ptr<at::Half>()),
                in_map_ptr, out_map_ptr
          );
  }
  else{
    // {\delta{out_feats}} X W^T = {\delta{in_feats}}
    _fgms_fusion_tf32_W_transpose<32, 4, 8, 16, 8, 16, 4, 2, 2>
              <<<dim3(DIV_UP(in_channel, 32), DIV_UP(sum_nnz, 128), 1), dim3(8, 32, 1)>>>(
                kpos_ptr, qkpos_ptr, k_vol, in_channel, out_channel, 
                out_feats_grad.data_ptr<float>(), 
                kernel.data_ptr<float>(), 
                in_feats_grad.data_ptr<float>(), 
                out_map_ptr, in_map_ptr
          );
    // in_feats^T X {\delta{out_feats}} = {\delta{W}}
    _fgms_fusion_tf32_I_transpose<32, 8, 8, 16, 8, 16, 2, 2, 1>
              <<<dim3(DIV_UP(sum_nnz, 128)), dim3(8, 16, 1)>>>(
                kpos_ptr, qkpos_ptr, k_vol, in_channel, out_channel, 
                in_feats.data_ptr<float>(), 
                out_feats_grad.data_ptr<float>(),
                kernel_grad.data_ptr<float>(),
                in_map_ptr, out_map_ptr
          );
  }
}